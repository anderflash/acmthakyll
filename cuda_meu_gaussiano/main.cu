#include "hip/hip_runtime.h"
#include <stdio.h>
#include <magick/MagickCore.h>
#include <andercamera.h>

__global__
void brilho(Quantum * pixels, int columns, int rows)
{

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= columns || y >= rows) return;
	int i = y * columns + x;

  int incremento = 100 * (QuantumRange+1)/256;
	if(i < rows*columns)
	{
	  int j = 4*i;
		pixels[j  ] = min(pixels[j  ]+incremento, QuantumRange);
		pixels[j+1] = min(pixels[j+1]+incremento, QuantumRange);
		pixels[j+2] = min(pixels[j+2]+incremento, QuantumRange);
		pixels[j+3] = min(pixels[j+3]+incremento, QuantumRange);
	}
}

__global__
void gaussiano(Quantum * pixels, int columns, int rows)
{
  /*Quantum * filtro[] = {0,2,0,2,2,2,0,2,0};
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= columns || y >= rows) return;
	int i = y * columns + x;
  int i = y * 
*/
}

__global__
void separar(const unsigned short* colormap, unsigned short* red, unsigned short* green, unsigned short* blue, int rows, int columns)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = y*columns+x;
	int j = 4*i;
	if(i < rows*columns)
	red[i] = min(colormap[j] + 10,255);
	green[i] = min(colormap[j+1] + 10, 255);
	blue[i] = min(colormap[j+2] + 10, 255);
}

int main(int argc, char**argv)
{
	Camera camera;
	ANDERopen_device(&camera);
	ANDERinit_device(&camera);
	ANDERstart_capturing(&camera);
	
	ExceptionInfo *exception;
	Image *image;
	ImageInfo *image_info;

  // Lendo a imagem
	MagickCoreGenesis(*argv, MagickTrue);          // Inicializando Magick
	exception = AcquireExceptionInfo();            // Criando controle de exceção
	image_info = CloneImageInfo((ImageInfo*) NULL);// Criando objeto de info da imagem
	strcpy(image_info->filename, "imagem.jpg");
	image = ReadImage(image_info, exception);      // Lendo a imagem e preenchendo a info e a exceção

  // Tratando exceção
	if(exception->severity != UndefinedException)  // Se houver exceção
		CatchException(exception);                   // Mostre a exceção
	if(image == (Image*) NULL)                     // Se não conseguir carregar a imagem
	{
		exit(1);                                     // Feche
	}
	
	void* p;
	int size;
	
	ANDERread_frame(&camera,&p,&size);
	

	// Exibindo a imagem

	//DisplayImages(image_info, image);

	PixelPacket* pixels = GetAuthenticPixels(image, 0, 0, image->columns, image->rows, exception);
	
	printf("Comeco\n");
	
	Quantum* d_pixels;
	int tamanho = image->columns * image->rows * sizeof(Quantum);

	hipMalloc(&d_pixels, tamanho*4);

	hipMemcpy(d_pixels, pixels, tamanho*4, hipMemcpyHostToDevice);
	
  dim3 blockSize(32,32,1);
	dim3 gridSize((image->columns + blockSize.x-1)/blockSize.x,(image->rows + blockSize.y-1)/blockSize.y,1);

	printf("Função\n");

	brilho<<<gridSize, blockSize>>>(d_pixels, image->columns, image->rows);

	hipMemcpy(pixels, d_pixels, tamanho*4, hipMemcpyDeviceToHost);

	/*Image* image2 = BlobToImage(image_info, blob, length, exception);

	if(exception->severity != UndefinedException)  // Se houver exceção
		CatchException(exception); 
	if(image2 == (Image*) NULL)
	{
		exit(1);
	}
	
	printf("Final\n");
	for(int i = 0; i < 10; i++)
	{
		printf("%d\n",blob[i]);
	}*/

	SyncAuthenticPixels(image,exception);

	DisplayImages(image_info,image);
	


	
	return 0;

}
